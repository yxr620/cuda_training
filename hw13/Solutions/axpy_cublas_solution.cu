
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

#define N 500000

// Simple short kernels
__global__
void kernel_a(float* x, float* y){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) y[idx] += 1;
}

__global__
void kernel_c(float* x, float* y){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) y[idx] += 1;
}

int main(){

hipStream_t stream1;

hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);

hipblasHandle_t cublas_handle;
hipblasCreate(&cublas_handle);
hipblasSetStream(cublas_handle, stream1);

// Set up host data and initialize
float* h_x;
float* h_y;

h_x = (float*) malloc(N * sizeof(float));
h_y = (float*) malloc(N * sizeof(float));

for (int i = 0; i < N; ++i){
    h_x[i] = float(i);
    h_y[i] = float(i);
}

// Print out the first 25 values of h_y
for (int i = 0; i < 25; ++i){
    printf("%2.0f ", h_y[i]);
}
printf("\n");

// Set up device data
float* d_x;
float* d_y;
float d_a = 5.0;

hipMalloc((void**) &d_x, N * sizeof(float));
hipMalloc((void**) &d_y, N * sizeof(float));
cudaCheckErrors("hipMalloc failed");

hipblasSetVector(N, sizeof(h_x[0]), h_x, 1, d_x, 1); // similar to cudaMemcpyHtoD
hipblasSetVector(N, sizeof(h_y[0]), h_y, 1, d_y, 1); // similar to cudaMemcpyHtoD
cudaCheckErrors("hipblasSetVector failed");

// Set up graph
hipGraph_t graph; // main graph
hipGraph_t libraryGraph; // sub graph for cuBLAS call
std::vector<hipGraphNode_t> nodeDependencies;
hipGraphNode_t kernelNode1, kernelNode2, libraryNode;

hipKernelNodeParams kernelNode1Params {0};
hipKernelNodeParams kernelNode2Params {0};

hipGraphCreate(&graph, 0); // create the graph
cudaCheckErrors("hipGraphCreate failure");

// kernel_a and kernel_c use same args
void *kernelArgs[2] = {(void *)&d_x, (void *)&d_y};

int threads = 512;
int blocks = (N + (threads - 1) / threads);

// Adding 1st node, kernel_a, as head node of graph
kernelNode1Params.func = (void *)kernel_a;
kernelNode1Params.gridDim = dim3(blocks, 1, 1);
kernelNode1Params.blockDim = dim3(threads, 1, 1);
kernelNode1Params.sharedMemBytes = 0;
kernelNode1Params.kernelParams = (void **)kernelArgs;
kernelNode1Params.extra = NULL;

hipGraphAddKernelNode(&kernelNode1, graph, NULL,
                         0, &kernelNode1Params);
cudaCheckErrors("Adding kernelNode1 failed");

nodeDependencies.push_back(kernelNode1); // manage dependecy vector

// Adding 2nd node, libraryNode, with kernelNode1 as dependency
hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal);
cudaCheckErrors("Stream capture begin failure");

// Library call
hipblasSaxpy(cublas_handle, N, &d_a, d_x, 1, d_y, 1);
cudaCheckErrors("hipblasSaxpy failure");

hipStreamEndCapture(stream1, &libraryGraph);
cudaCheckErrors("Stream capture end failure");

hipGraphAddChildGraphNode(&libraryNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), libraryGraph);
cudaCheckErrors("Adding libraryNode failed");

nodeDependencies.clear();
nodeDependencies.push_back(libraryNode); // manage dependency vector

// Adding 3rd node, kernel_c, with libraryNode as dependency
kernelNode2Params.func = (void *)kernel_c;
kernelNode2Params.gridDim = dim3(blocks, 1, 1);
kernelNode2Params.blockDim = dim3(threads, 1, 1);
kernelNode2Params.sharedMemBytes = 0;
kernelNode2Params.kernelParams = (void **)kernelArgs;
kernelNode2Params.extra = NULL;

hipGraphAddKernelNode(&kernelNode2, graph, nodeDependencies.data(),
                         nodeDependencies.size(), &kernelNode2Params);
cudaCheckErrors("Adding kernelNode2 failed");

nodeDependencies.clear();
nodeDependencies.push_back(kernelNode2); // manage dependency vector

hipGraphNode_t *nodes = NULL;
size_t numNodes = 0;
hipGraphGetNodes(graph, nodes, &numNodes);
cudaCheckErrors("Graph get nodes failed");
printf("Number of the nodes in the graph = %zu\n", numNodes);

hipGraphExec_t instance;
hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
cudaCheckErrors("Graph instantiation failed");

// Launch the graph instance 100 times
for (int i = 0; i < 100; ++i){
    hipGraphLaunch(instance, stream1);
    hipStreamSynchronize(stream1);
}
cudaCheckErrors("Graph launch failed");
hipDeviceSynchronize();

// Copy memory back to host
hipMemcpy(h_y, d_y, N, hipMemcpyDeviceToHost);
cudaCheckErrors("Finishing memcpy failed");

hipDeviceSynchronize();

// Print out the first 25 values of h_y
for (int i = 0; i < 25; ++i){
    printf("%2.0f ", h_y[i]);
}
printf("\n");

return 0;

}
