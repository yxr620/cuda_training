
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

#define N 500000

// Simple short kernels
__global__
void kernel_a(float * x, float * y){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = 2.0*x[idx] + y[idx];

}

__global__
void kernel_b(float * x, float * y){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = 2.0*x[idx] + y[idx];

}

__global__
void kernel_c(float * x, float * y){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = 2.0*x[idx] + y[idx];

}

__global__
void kernel_d(float * x, float * y){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = 2.0*x[idx] + y[idx];

}

int main(){

// Set up and create events
hipEvent_t event1;
hipEvent_t event2;

hipEventCreateWithFlags(&event1, hipEventDisableTiming);
hipEventCreateWithFlags(&event2, hipEventDisableTiming);

// Set up and create streams
const int num_streams = 2;

hipStream_t streams[num_streams];

for (int i = 0; i < num_streams; ++i){
    hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
}

// Set up and initialize host data
float* h_x;
float* h_y;

h_x = (float*) malloc(N * sizeof(float));
h_y = (float*) malloc(N * sizeof(float));

for (int i = 0; i < N; ++i){
    h_x[i] = (float)i;
    h_y[i] = (float)i;
//    printf("%2.0f ", h_x[i]);
}
printf("\n");

// Set up device data
float* d_x;
float* d_y;

hipMalloc((void**) &d_x, N * sizeof(float));
hipMalloc((void**) &d_y, N * sizeof(float));
cudaCheckErrors("hipMalloc failed");

hipMemcpy(d_x, h_x, N, hipMemcpyHostToDevice);
hipMemcpy(d_y, h_y, N, hipMemcpyHostToDevice);
cudaCheckErrors("hipMalloc failed");

int threads = 512;
int blocks = (N + (threads - 1) / threads);

// Launching work
for (int i = 0; i < 100; ++i){
    kernel_a<<<blocks, threads, 0, streams[0]>>>(d_x, d_y);
    cudaCheckErrors("Kernel a failed");

    hipEventRecord(event1, streams[0]);
    cudaCheckErrors("Event record failed");

    kernel_b<<<blocks, threads, 0, streams[0]>>>(d_x, d_y);
    cudaCheckErrors("Kernel b failed");

    hipStreamWaitEvent(streams[1], event1);
    cudaCheckErrors("Event wait failed");

    kernel_c<<<blocks, threads, 0, streams[1]>>>(d_x, d_y);
    cudaCheckErrors("Kernel c failed");

    hipEventRecord(event2, streams[1]);
    cudaCheckErrors("Event record failed");

    hipStreamWaitEvent(streams[0], event2);
    cudaCheckErrors("Event wait failed");

    kernel_d<<<blocks, threads, 0, streams[0]>>>(d_x, d_y);
    cudaCheckErrors("Kernel d failed");

    hipStreamSynchronize(streams[0]);
}

// Copy data back to host
hipMemcpy(h_y, d_y, N, hipMemcpyDeviceToHost);
cudaCheckErrors("Finishing memcpy failed");

hipDeviceSynchronize();

// Print out the first 25 values of h_y
for (int i = 0; i < 25; ++i){
    printf("%2.0f ", h_y[i]);
}
printf("\n");

return 0;
}
