
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cstdio>

// alternating harmonic series: https://en.wikipedia.org/wiki/Harmonic_series_(mathematics)#Alternating_harmonic_series
// compute alternating harmonic series member based on index n
__device__ auto ahs(size_t n){ return ((n&1)?1:-1)/(double)n;}

// blocksize must be a power of 2, less than or equal to 1024
#define BLOCK_SIZE 512

// estimate summation of alternating harmonic series
template <typename T>
__global__ void estimate_sum_ahs(size_t length, T *sum){
  __shared__ T smem[BLOCK_SIZE];
  size_t idx = blockDim.x*blockIdx.x+threadIdx.x;
  smem[threadIdx.x] = (idx < length)?ahs(idx):0;

  for (int i = blockDim.x>>1; i > 0; i >>= 1){
    __syncthreads();
    if (threadIdx.x < i) smem[threadIdx.x] += smem[threadIdx.x+i];}

  if (threadIdx.x == 0) atomicAdd(sum, smem[0]);
}

typedef double ft;

int main(int argc, char* argv[]){
  size_t my_length = 1048576; // allow user to override default estimation length with command-line argument
  if (argc > 1) my_length = atol(argv[1]);
  ft *sum;
  hipError_t err = hipMallocManaged(&sum, sizeof(ft));
  if (err != hipSuccess) {std::cout << "Error: " << hipGetErrorString(err) << std::endl; return 0;}
  *sum = 0;
  dim3 block(BLOCK_SIZE);
  dim3 grid((my_length+block.x-1)/block.x);
  estimate_sum_ahs<<<grid, block>>>(my_length, sum);
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {std::cout << "Error: " << hipGetErrorString(err) << std::endl; return 0;}
  std::cout << "Estimated value: " << *sum << " Expected value: "  << log(2)  << std::endl;
  return 0;
}
