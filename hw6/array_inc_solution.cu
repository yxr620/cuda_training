#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

template <typename T>
void alloc_bytes(T &ptr, size_t num_bytes){

  hipMallocManaged(&ptr, num_bytes);
}

__global__ void inc(int *array, size_t n){
  size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
  while (idx < n){
    array[idx]++;
    idx += blockDim.x*gridDim.x; // grid-stride loop
    }
}

const size_t  ds = 32ULL*1024ULL*1024ULL;

int main(){

  int *h_array;
  alloc_bytes(h_array, ds*sizeof(h_array[0]));
  cudaCheckErrors("hipMallocManaged Error");
  memset(h_array, 0, ds*sizeof(h_array[0]));
  hipMemPrefetchAsync(h_array, ds*sizeof(h_array[0]), 0); // add in step 2c
  inc<<<256, 256>>>(h_array, ds);
  cudaCheckErrors("kernel launch error");
  hipMemPrefetchAsync(h_array, ds*sizeof(h_array[0]), hipCpuDeviceId); // add in step 2c
  hipDeviceSynchronize();
  cudaCheckErrors("kernel execution error");
  for (int i = 0; i < ds; i++) 
    if (h_array[i] != 1) {printf("mismatch at %d, was: %d, expected: %d\n", i, h_array[i], 1); return -1;}
  printf("success!\n"); 
  return 0;
}
